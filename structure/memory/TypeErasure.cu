#include "structure/memory/TypeErasure.cuh"

#include <hip/hip_runtime.h>
#include "util/cuda_utils.cuh"

namespace bitgraph {
    namespace memory {
        
        void* TypeErasedVector::alloc(size_t N) {
            size_t dtype_size = gremlinxx::comparison::C_size[this->dtype];

            switch(this->mem_type) {
                case bitgraph::memory::memory_type::HOST: {
                    return static_cast<void*>(new char[N * dtype_size]);
                }
                case bitgraph::memory::memory_type::DEVICE: {
                    void* ptr;
                    hipMalloc(&ptr, dtype_size * N);
                    hipDeviceSynchronize();
                    cudaCheckErrors("TypeErasedVector alloc device memory");
                    return ptr;
                }
                case bitgraph::memory::memory_type::MANAGED: {
                    void* ptr;
                    hipMallocManaged(&ptr, dtype_size * N);
                    hipDeviceSynchronize();
                    cudaCheckErrors("TypeErasedVector alloc managed memory");
                    return ptr;
                }
                case bitgraph::memory::memory_type::PINNED: {
                    void* ptr;
                    hipHostMalloc(&ptr, dtype_size * N);
                    hipDeviceSynchronize();
                    cudaCheckErrors("TypeErasedVector alloc pinned memory");
                    return ptr;
                }
            }

            throw std::runtime_error("Invalid memory type provided to TypeErasedVector alloc()");
        }

        void TypeErasedVector::dealloc(void* ptr) {
            switch(this->mem_type) {
                case bitgraph::memory::memory_type::HOST: {
                    delete static_cast<char*>(ptr);
                    return;
                }
                case bitgraph::memory::memory_type::MANAGED: {
                    hipFree(ptr);
                    hipDeviceSynchronize();
                    cudaCheckErrors("TypeErasedVector dealloc managed memory");
                    return;
                }
                case bitgraph::memory::memory_type::DEVICE: {
                    hipFree(ptr);
                    hipDeviceSynchronize();
                    cudaCheckErrors("TypeErasedVector dealloc device memory");
                    return;
                }
                case bitgraph::memory::memory_type::PINNED: {
                    hipHostFree(ptr);
                    hipDeviceSynchronize();
                    cudaCheckErrors("TypeErasedVector dealloc pinned memory");
                    return;
                }
            }

            throw std::runtime_error("Invalid memory type provided to TypeErasedVector dealloc");
        }

        // Copies from src (first arg) to dst (second arg) using hipMemcpy.
        void TypeErasedVector::copy(void* src, void* dst, size_t size) {
            hipMemcpy(dst, src, gremlinxx::comparison::C_size[this->dtype] * size, hipMemcpyDefault);
            cudaCheckErrors("TypeErasedVector copy");
        }

        // Creates a blank vector with the given memory type and data type.
        TypeErasedVector::TypeErasedVector(bitgraph::memory::memory_type mem_type, gremlinxx::comparison::C dtype) {
            this->mem_type = mem_type;
            this->dtype = dtype;
            this->filled_size = 0;
            this->reserved_size = 0;
            this->data_ptr = nullptr;
            this->view = false;
        }

        // Default constructor; creates a blank device vector of FLOAT64 dtype
        TypeErasedVector::TypeErasedVector()
        : TypeErasedVector(bitgraph::memory::memory_type::DEVICE, gremlinxx::comparison::C::FLOAT64) {}

        // Creates a vector of size N unitialized values of the given data type and given memory type.
        TypeErasedVector::TypeErasedVector(bitgraph::memory::memory_type mem_type, gremlinxx::comparison::C dtype, size_t N) {
            this->mem_type = mem_type;
            this->dtype = dtype;
            this->filled_size = 0;
            this->reserved_size = 0;
            this->data_ptr = nullptr;
            this->view = false;

            this->resize(N);
        }

        // Creates a vector corresponding to the provided data.  If view=true then this vector is only a view
        // over the provided data.  If view=false then this vector will own a copy of the provided data.
        TypeErasedVector::TypeErasedVector(bitgraph::memory::memory_type mem_type, gremlinxx::comparison::C dtype, void* data, size_t N, bool view) {
            this->mem_type = mem_type;
            this->dtype = dtype;
            this->view = view;
            this->reserved_size = 0;

            if(this->view) { 
                this->data_ptr = data; 
                this->filled_size = N;
                this->reserved_size = N;
            }
            else {
                this->resize(N);
                this->copy(data, this->data_ptr, N);
            }
        }

        TypeErasedVector::TypeErasedVector(TypeErasedVector& orig) {
            this->mem_type = orig.mem_type;
            this->dtype = orig.dtype;
            this->filled_size = orig.filled_size;
            this->reserved_size = 0;
            this->view = false;

            this->resize(orig.filled_size);
            this->copy(orig.data_ptr, this->data_ptr, orig.filled_size);                    
        }

        TypeErasedVector::~TypeErasedVector() {
            if(this->data_ptr != nullptr && !this->view) {
                this->dealloc(this->data_ptr);
            }
        }

        TypeErasedVector::TypeErasedVector(TypeErasedVector&& other) noexcept {
            this->data_ptr = std::move(other.data_ptr);
            this->filled_size = std::move(other.filled_size);
            this->reserved_size = std::move(other.reserved_size);
            this->dtype = std::move(other.dtype);
            this->mem_type = std::move(other.mem_type);
            this->view = other.view;
            other.view = true;
        }
        
        TypeErasedVector& TypeErasedVector::operator=(TypeErasedVector&& other) noexcept {
            this->data_ptr = std::move(other.data_ptr);
            this->filled_size = std::move(other.filled_size);
            this->reserved_size = std::move(other.reserved_size);
            this->dtype = std::move(other.dtype);
            this->mem_type = std::move(other.mem_type);
            this->view = other.view;
            other.view = true;

            return *this;
        }

        void TypeErasedVector::push_back() {
            throw std::runtime_error("push_back unimplemented");
        }

        void TypeErasedVector::reserve(size_t N) {
            throw std::runtime_error("reserve unimplemented");
        }

        void TypeErasedVector::insert() {
            throw std::runtime_error("insert unimplemented");
        }

        void TypeErasedVector::insert(size_t ix_start, TypeErasedVector& new_elements) {
            if(this->view) throw std::runtime_error("Cannot insert into a view!");
            if(this->dtype != new_elements.dtype) throw std::runtime_error("Data type of inserting vector must match!");

            size_t old_size = this->size();
            size_t new_size = old_size + new_elements.size();
            
            void* new_data = this->data_ptr;
            std::cout << "new size: " << new_size << std::endl;
            std::cout << "reserved size: " << reserved_size << std::endl;
            if(new_size > reserved_size) {
                new_data = this->alloc(new_size);
                this->reserved_size = new_size;   
            }

            size_t elements_to_copy = old_size - ix_start;
            size_t element_size = gremlinxx::comparison::C_size[this->dtype];

            if(elements_to_copy > 0) {
                this->copy(
                    static_cast<char*>(this->data_ptr) + (element_size * ix_start),
                    static_cast<char*>(new_data) + (element_size * (ix_start + new_elements.size())),
                    elements_to_copy
                );
            }

            this->copy(
                new_elements.data(),
                static_cast<char*>(new_data) + (element_size * ix_start),
                new_elements.size()
            );

            if(this->data_ptr != new_data) {
                if(ix_start > 0) {
                    this->copy(
                        this->data_ptr,
                        new_data,
                        ix_start
                    );
                }

                cudaCheckErrors("check errors before dealloc");
                this->dealloc(this->data_ptr);
                this->data_ptr = new_data;
            }

            this->filled_size = new_size;
        }

        /*
            Copies the vector to the host and prints it.
        */
        void TypeErasedVector::print() {
            std::vector<size_t> h_data(this->size());
            hipMemcpy(h_data.data(), this->data(), gremlinxx::comparison::C_size[this->dtype] * this->size(), hipMemcpyDefault);
            hipDeviceSynchronize();
            cudaCheckErrors("copy to host");
            for(auto x : h_data) std::cout << x << " ";
            std::cout << std::endl;
        }

        void TypeErasedVector::resize(size_t N) {
            if(this->view) throw std::runtime_error("Cannot resize a view!");

            bool empty = (this->reserved_size == 0);
            
            // Don't resize if there is already enough space reserved
            if(N < reserved_size) {
                this->filled_size = N;
                return;
            }

            void* new_data = this->alloc(N);
            if(!empty) {
                this->copy(this->data_ptr, new_data, this->filled_size);
                this->dealloc(this->data_ptr);
            }
            
            this->data_ptr = new_data;
            this->filled_size = N;
            this->reserved_size = N;
        }

        TypeErasedVector make_vector_like(TypeErasedVector& other, size_t N) {
            return TypeErasedVector(
                other.get_mem_type(),
                other.get_dtype(),
                N
            );
        }

    }
}