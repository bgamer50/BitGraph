#include "step/gpu/GPUBindStep.cuh"

#include "step/gpu/GPUTraversalHelper.cuh"
#include "util/cuda_utils.cuh"

GPUBindStep::GPUBindStep(gremlinxx::comparison::C dtype)
: TraversalStep(MAP, GPU_BIND_STEP) {
    this->dtype = dtype;
}

GPUBindStep::GPUBindStep()
: TraversalStep(MAP, GPU_BIND_STEP) {
    this->dtype = gremlinxx::comparison::C::INT32;
}

std::string GPUBindStep::getInfo() {
    std::stringstream ss;
    ss << "GPUBindStep{" << gremlinxx::comparison::C_to_string[this->dtype] << "}";
    return ss.str();
}

void GPUBindStep::apply(GraphTraversal* parent_traversal, TraverserSet& traversers) {
    gpu_traverser_info_t traverser_info;
    traverser_info.traversers = C_TO_GPU(this->dtype, traversers);
    traverser_info.traverser_dtype = this->dtype;

    size_t* originating_traversers;
    hipMalloc((void**) &originating_traversers, sizeof(size_t) * traversers.size());
    hipDeviceSynchronize();
    cudaCheckErrors("allocate originating traversers");

    std::vector<size_t> h_originating_traversers(traversers.size());
    std::iota(h_originating_traversers.begin(), h_originating_traversers.end(), 0);
    hipMemcpy(originating_traversers, h_originating_traversers.data(), sizeof(size_t) * traversers.size(), hipMemcpyDefault);
    hipDeviceSynchronize();
    cudaCheckErrors("copy originating traversers");
    
    traverser_info.paths.push_back(std::make_pair(originating_traversers, traversers.size()));

    traverser_info.num_traversers = traversers.size();
    traverser_info.original_traversers.swap(traversers);

    traversers.push_back(Traverser(traverser_info));
}